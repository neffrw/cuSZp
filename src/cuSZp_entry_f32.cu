#include "cuSZp_entry_f32.h"
#include "cuSZp_f32.h"
#include <stdio.h>

void SZp_compress_hostptr_f32(float* oriData, unsigned char* cmpBytes, size_t nbEle, size_t* cmpSize, float errorBound)
{
    // Data blocking.
    int bsize = cmp_tblock_size_f32;
    int gsize = (nbEle + bsize * cmp_chunk_f32 - 1) / (bsize * cmp_chunk_f32);
    int cmpOffSize = gsize + 1;
    int pad_nbEle = gsize * bsize * cmp_chunk_f32;

    // Initializing global memory for GPU compression.
    float* d_oriData;
    unsigned char* d_cmpData;
    unsigned int* d_cmpOffset;
    int* d_flag;
    hipMalloc((void**)&d_oriData, sizeof(float)*pad_nbEle);
    hipMemcpy(d_oriData, oriData, sizeof(float)*pad_nbEle, hipMemcpyHostToDevice);
    hipMalloc((void**)&d_cmpData, sizeof(float)*pad_nbEle);
    hipMallocManaged((void**)&d_cmpOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_cmpOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_flag, sizeof(int)*cmpOffSize);
    hipMemset(d_flag, 0, sizeof(int)*cmpOffSize);
    hipMemset(d_oriData + nbEle, 0, (pad_nbEle - nbEle) * sizeof(float));

    // Initializing CUDA Stream.
    hipStream_t stream;
    hipStreamCreate(&stream);

    // cuSZp GPU compression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    SZp_compress_kernel_f32<<<gridSize, blockSize, 0, stream>>>(d_oriData, d_cmpData, d_cmpOffset, d_flag, errorBound, nbEle);
    hipDeviceSynchronize();

    // Obtain compression ratio and move data back to CPU.  
    *cmpSize = (size_t)d_cmpOffset[cmpOffSize-1] + (nbEle+31)/32;
    hipMemcpy(cmpBytes, d_cmpData, *cmpSize*sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Free memory that is used.
    hipFree(d_oriData);
    hipFree(d_cmpData);
    hipFree(d_cmpOffset);
    hipFree(d_flag);
    hipStreamDestroy(stream);
}


void SZp_decompress_hostptr_f32(float* decData, unsigned char* cmpBytes, size_t nbEle, size_t cmpSize, float errorBound)
{
    // Data blocking.
    int bsize = dec_tblock_size_f32;
    int gsize = (nbEle + bsize * dec_chunk_f32 - 1) / (bsize * dec_chunk_f32);
    int cmpOffSize = gsize + 1;
    int pad_nbEle = gsize * bsize * dec_chunk_f32;

    // Initializing global memory for GPU compression.
    float* d_decData;
    unsigned char* d_cmpData;
    unsigned int* d_cmpOffset;
    int* d_flag;
    hipMalloc((void**)&d_decData, sizeof(float)*pad_nbEle);
    hipMemset(d_decData, 0, sizeof(float)*pad_nbEle);
    hipMalloc((void**)&d_cmpData, sizeof(float)*pad_nbEle);
    hipMemcpy(d_cmpData, cmpBytes, sizeof(unsigned char)*cmpSize, hipMemcpyHostToDevice);
    hipMalloc((void**)&d_cmpOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_cmpOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_flag, sizeof(int)*cmpOffSize);
    hipMemset(d_flag, 0, sizeof(int)*cmpOffSize);

    // Initializing CUDA Stream.
    hipStream_t stream;
    hipStreamCreate(&stream);

    // cuSZp GPU compression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    SZp_decompress_kernel_f32<<<gridSize, blockSize, 0, stream>>>(d_decData, d_cmpData, d_cmpOffset, d_flag, errorBound, nbEle);
    hipDeviceSynchronize();

    // Move data back to CPU.
    hipMemcpy(decData, d_decData, sizeof(float)*nbEle, hipMemcpyDeviceToHost);

    // Free memoy that is used.
    hipFree(d_decData);
    hipFree(d_cmpData);
    hipFree(d_cmpOffset);
    hipFree(d_flag);
    hipStreamDestroy(stream);
}


void SZp_compress_deviceptr_f32(float* d_oriData, unsigned char* d_cmpBytes, size_t nbEle, size_t* cmpSize, float errorBound, hipStream_t stream)
{
    int bsize = cmp_tblock_size_f32;
    int gsize = (nbEle + bsize * cmp_chunk_f32 - 1) / (bsize * cmp_chunk_f32);
    int cmpOffSize = gsize + 1;
    int pad_nbEle = gsize * bsize * cmp_chunk_f32;

    // Initializing global memory for GPU compression.
    unsigned int* d_cmpOffset;
    int* d_flag;
    hipMallocManaged((void**)&d_cmpOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_cmpOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_flag, sizeof(int)*cmpOffSize);
    hipMemset(d_flag, 0, sizeof(int)*cmpOffSize);
    // hipMemset(d_oriData + nbEle, 0, (pad_nbEle - nbEle) * sizeof(float));

    // cuSZp GPU compression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    SZp_compress_kernel_f32<<<gridSize, blockSize, 0, stream>>>(d_oriData, d_cmpBytes, d_cmpOffset, d_flag, errorBound, nbEle);
    hipDeviceSynchronize();

    // Obtain compression ratio and move data back to CPU.  
    *cmpSize = (size_t)d_cmpOffset[cmpOffSize-1] + (nbEle+31)/32;

    // Free memory that is used.
    hipFree(d_cmpOffset);
    hipFree(d_flag);
}


void SZp_decompress_deviceptr_f32(float* d_decData, unsigned char* d_cmpBytes, size_t nbEle, size_t cmpSize, float errorBound, hipStream_t stream)
{
    // Data blocking.
    int bsize = dec_tblock_size_f32;
    int gsize = (nbEle + bsize * dec_chunk_f32 - 1) / (bsize * dec_chunk_f32);
    int cmpOffSize = gsize + 1;

    // Initializing global memory for GPU compression.
    unsigned int* d_cmpOffset;
    int* d_flag;
    hipMalloc((void**)&d_cmpOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_cmpOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_flag, sizeof(int)*cmpOffSize);
    hipMemset(d_flag, 0, sizeof(int)*cmpOffSize);

    // cuSZp GPU compression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    SZp_decompress_kernel_f32<<<gridSize, blockSize, 0, stream>>>(d_decData, d_cmpBytes, d_cmpOffset, d_flag, errorBound, nbEle);
    hipDeviceSynchronize();

    // Free memoy that is used.
    hipFree(d_cmpOffset);
    hipFree(d_flag);
}